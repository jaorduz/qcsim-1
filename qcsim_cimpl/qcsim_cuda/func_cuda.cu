#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <math.h>

extern "C" {
#include "func.h"
#include "random.h"
}
#include "common_cuda.h"

/*
n-qubit non-unitary operation
initialize all qubits
*/
__global__ void kernel_op_init(double* nstate, const size_t dim) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		nstate[2 * i] = 0.;
		nstate[2 * i + 1] = 0.;
		if (i == 0) nstate[2 * i] = 1.;

		i += blockDim.x * gridDim.x;
	}
}
void op_init(double* nstate, const size_t dim) {
	unsigned int blockCount, threadCount;
	threadCount = min((unsigned int)dim,g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim/g_maxThreadsPerBlock,1);

	kernel_op_init << < blockCount, threadCount >> > (nstate, dim);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { 
		fprintf(stderr, "cuda op_init failed : %s\n", hipGetErrorString(cudaStatus)); 
	}
}



/*
1qubit unitary operation
u1,u2,u3 is equivalnent to U(\theta,\phi,\lambda) in QASM
*/
__global__ void kernel_op_u(const double *state, double* nstate, const size_t dim, const size_t targetMask,
	const double u00r, const double u00i, const double u01r, const double u01i, const double u10r, const double u10i, const double u11r, const double u11i) {

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tmp;
	while (i < dim) {
		tmp = i^targetMask;
		if ((i&targetMask) == 0) {
			nstate[2 * i] = u00r * state[2 * i] - u00i * state[2 * i + 1] + u01r * state[2 * tmp] - u01i * state[2 * tmp + 1];
			nstate[2 * i + 1] = u00r * state[2 * i + 1] + u00i * state[2 * i] + u01r * state[2 * tmp + 1] + u01i * state[2 * tmp];
		}
		else {
			nstate[2 * i] = u10r * state[2 * tmp] - u10i * state[2 * tmp + 1] + u11r * state[2 * i] - u11i * state[2 * i + 1];
			nstate[2 * i + 1] = u10r * state[2 * tmp + 1] + u10i * state[2 * tmp] + u11r * state[2 * i + 1] + u11i * state[2 * i];
		}

		i += blockDim.x * gridDim.x;
	}
}
void op_u(const double* state, double* nstate, const size_t dim, const unsigned int target, const double u1, const double u2, const double u3) {
	const size_t targetMask = ((size_t)1) << target;
	double u00r, u01r, u10r, u11r, u00i, u01i, u10i, u11i;
	unsigned int blockCount, threadCount;

	u00r = cos((u2 + u3) / 2) * cos(u1 / 2);
	u00i = -sin((u2 + u3) / 2) * cos(u1 / 2);
	u01r = -cos((u2 - u3) / 2) * sin(u1 / 2);
	u01i = sin((u2 - u3) / 2) * sin(u1 / 2);
	u10r = cos((u2 - u3) / 2) * sin(u1 / 2);
	u10i = sin((u2 - u3) / 2) * sin(u1 / 2);
	u11r = cos((u2 + u3) / 2) * cos(u1 / 2);
	u11i = sin((u2 + u3) / 2) * cos(u1 / 2);

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_u << < blockCount, threadCount >> > (state, nstate, dim, targetMask, u00r, u00i, u01r, u01i, u10r, u10i, u11r, u11i);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_u failed : %s\n", hipGetErrorString(cudaStatus));
	}
}


/*
2qubit unitary operation
control not

"target" must be different from "control"
*/
__global__ void kernel_op_cx(const double *state, double* nstate, const size_t dim, const size_t targetMask, const size_t controlMask) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int tmp;
	while (i < dim) {
		if (i&controlMask) {
			tmp = i^targetMask;
			nstate[2 * i] = state[2 * tmp];
			nstate[2 * i + 1] = state[2 * tmp + 1];
		}
		else {
			nstate[2 * i] = state[2 * i];
			nstate[2 * i + 1] = state[2 * i + 1];
		}

		i += blockDim.x * gridDim.x;
	}
}
void op_cx(const double* state, double* nstate, const size_t dim, const unsigned int target, const unsigned int control) {
	const size_t targetMask = ((size_t)1) << target;
	const size_t controlMask = ((size_t)1) << control;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_cx << < blockCount, threadCount >> > (state, nstate, dim, targetMask, controlMask);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_cx failed : %s\n", hipGetErrorString(cudaStatus));
	}
}


/*
1qubit non-unitary operation
post-select 0-outcome
*/
__global__ void kernel_op_post0(const double *state, double* nstate, const size_t dim, const size_t targetMask, const double norm) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		if ((i&targetMask) == 0) {
			nstate[2 * i] = state[2 * i] * norm;
			nstate[2 * i + 1] = state[2 * i + 1] * norm;
		}
		else {
			nstate[2 * i] = 0;
			nstate[2 * i + 1] = 0;
		}

		i += blockDim.x * gridDim.x;
	}
}

void op_post0(const double* state, double* nstate, const size_t dim, const unsigned int target, const double norm) {
	const size_t targetMask = ((size_t)1) << target;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_post0 << <blockCount, threadCount >> > (state, nstate, dim, targetMask, norm);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_post0 failed : %s\n", hipGetErrorString(cudaStatus));
	}
}

/*
1qubit non-unitary operation
post-select 1-outcome
*/
__global__ void kernel_op_post1(const double *state, double* nstate, const size_t dim, const size_t targetMask, const double norm) {

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		if (i&targetMask) {
			nstate[2 * i] = state[2 * i] * norm;
			nstate[2 * i + 1] = state[2 * i + 1] * norm;
		}
		else {
			nstate[2 * i] = 0;
			nstate[2 * i + 1] = 0;
		}

		i += blockDim.x * gridDim.x;
	}
}
void op_post1(const double* state, double* nstate, const size_t dim, const unsigned int target, const double norm) {
	const size_t targetMask = ((size_t)1) << target;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_post1 << <blockCount, threadCount >> > (state, nstate, dim, targetMask, norm);
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_post1 failed : %s\n", hipGetErrorString(cudaStatus));
	}
}

/*
1qubit non-unitary operation
measurement, and return outcome
*/
unsigned int op_meas(const double* state, double* nstate, const size_t dim, const unsigned int target) {
	double prob1;
	double randomValue;
	double norm;
	unsigned int outcome;

	prob1 = stat_prob1(state, nstate, dim, target);
	randomValue = rng();
	if (randomValue > prob1) {
		outcome = 0;
		norm = 1. / sqrt(1 - prob1);
		op_post0(state, nstate, dim, target, norm);
	}
	else {
		outcome = 1;
		norm = 1. / sqrt(prob1);
		op_post1(state, nstate, dim, target, norm);
	}
	return outcome;
}

/*
calculate probability with which we obtain outcome 1
*/
template <unsigned int blockSize>
__global__ void kernel_stat_prob1_optimized(const double *g_idata, double *g_odata, const size_t N, const size_t targetMask)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < N) {
		if((i/2)&targetMask)				
			sdata[tid] += g_idata[i] * g_idata[i];
		if(((i+blockSize)/2)&targetMask)	
			sdata[tid] += g_idata[i + blockSize] * g_idata[i + blockSize];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 1024){ if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
template <unsigned int blockSize>
__global__ void kernel_stat_sum_optimized(const double *g_idata, double *g_odata, const size_t N)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < N) {
		sdata[tid] += g_idata[i] + g_idata[i + blockSize];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void kernel_stat_prob1(const double *g_idata, double *g_odata, const size_t N, const size_t targetMask)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = ((i/2)&targetMask) ? g_idata[i] * g_idata[i] : 0.;
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void kernel_stat_sum(const double *g_idata, double *g_odata, const size_t N)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i]; 
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

double stat_prob1(const double* state, double* workspace, const size_t dim, const unsigned int target) {
	const size_t targetMask = ((size_t)1) << target;
	double prob1 = 0.;
	unsigned int threadCount, blockCount, sharedMemSize, cursor, N;
	hipError_t cudaStatus;

	N = (unsigned int)dim * 2;
	threadCount = min(N, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)N / (2*g_maxThreadsPerBlock), 1);
	sharedMemSize = threadCount * sizeof(double);

	// mapping squared values from state to workspace 
	if (threadCount == 1024 && blockCount>1)	kernel_stat_prob1_optimized<1024> << <blockCount, threadCount, sharedMemSize >> > (state, workspace, N, targetMask);
	else										kernel_stat_prob1 << <blockCount, threadCount, sharedMemSize >> > (state, workspace, N, targetMask);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "stat_prob1 fail : %s\n", hipGetErrorString(cudaStatus));
	}

	// accumulate workspace with length = blockCount
	cursor = 0;
	N = blockCount;
	while(N>1) {
		threadCount = min(N, g_maxThreadsPerBlock);
		blockCount = max((unsigned int)N / (2 * g_maxThreadsPerBlock), 1);
		sharedMemSize = threadCount * sizeof(double);

		double* fromPtr = workspace + cursor;
		double* toPtr = workspace + cursor + N;
		if (threadCount == 1024 && blockCount > 1)	kernel_stat_sum_optimized<1024> << <blockCount, threadCount, sharedMemSize >> > (fromPtr, toPtr , N);
		else										kernel_stat_sum << <blockCount, threadCount, sharedMemSize >> > (fromPtr, toPtr, N);
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "stat_prob1 loop fail : %s\n", hipGetErrorString(cudaStatus));
		}

		cursor += N;
		N = blockCount;
	}

	cudaStatus = hipMemcpy(&prob1, workspace + cursor, sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy fail : %s\n", hipGetErrorString(cudaStatus));
	}
	//printf("prob1 : %lf\n", prob1);
	return prob1;
}

void dump_vector(const double* state, const size_t dim, FILE* outStream) {
	size_t i; 
	double norm = 0.;
	double* local = (double*)malloc(2 * dim * sizeof(double));
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(local,state,2*dim*sizeof(double),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}
	else {
		for (i = 0; i < dim; i++) {
			fprintf(outStream, "%lld : %lf , %lf\n", i, local[2 * i], local[2 * i + 1]);
			norm += local[2 * i] * local[2 * i] + local[2 * i + 1] * local[2 * i + 1];
		}
		printf("norm :%lf\n", norm);
	}
	free(local);
}