
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

extern "C" {
#include "allocate.h"
}
#include "common_cuda.h"

unsigned int g_maxThreadsPerBlock;
unsigned int g_maxBlocksPerGrid;

int initDevice() {
	hipError_t  cudaStatus;

	hipDeviceProp_t deviceProp;
	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGetDeviceProperties failed : %s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	g_maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
	g_maxBlocksPerGrid = deviceProp.maxGridSize[0];

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed : %s\n", hipGetErrorString(cudaStatus));
		return -1;
	}
	return 0;
}

double* stateAllocate(const int n) {
	hipError_t  cudaStatus;
	size_t dim = ((size_t)1) << n;
	if (dim >= g_maxThreadsPerBlock * g_maxBlocksPerGrid) {
		fprintf(stderr, "Too many elements : %lld elements required, but allowed size are %ld * %ld \n", dim, g_maxThreadsPerBlock, g_maxBlocksPerGrid);
		return NULL;
	}

	double* ptr;
	cudaStatus = hipMalloc((void**)&ptr, 2*dim * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed : %s\n", hipGetErrorString(cudaStatus));
		return NULL;
	}
	return ptr;
}

void stateRelease(double* state) {
	hipFree(state);
}

void closeDevice() {
}