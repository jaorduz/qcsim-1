#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <math.h>

extern "C" {
#include "func.h"
#include "random.h"
}
#include "common_cuda.h"

#define MIN(p,q) (p<q?p:q)
#define MAX(p,q) (p>q?p:q)

/*
n-qubit non-unitary operation
initialize all qubits
*/
__global__ void kernel_op_init(double* nstate, const size_t dim) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		nstate[2 * i] = 0.;
		nstate[2 * i + 1] = 0.;
		if (i == 0) nstate[2 * i] = 1.;

		i += blockDim.x * gridDim.x;
	}
}
void op_init(double* nstate, const size_t dim) {
	unsigned int blockCount, threadCount;
	threadCount = min((unsigned int)dim,g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim/g_maxThreadsPerBlock,1);

	kernel_op_init << < blockCount, threadCount >> > (nstate, dim);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { 
		fprintf(stderr, "cuda op_init failed : %s\n", hipGetErrorString(cudaStatus)); 
	}
}



/*
1qubit unitary operation
u1,u2,u3 is equivalnent to U(\theta,\phi,\lambda) in QASM
*/
__global__ void kernel_op_u(double *state, const size_t dim, const size_t targetMask,
	const double u00r, const double u00i, const double u01r, const double u01i, const double u10r, const double u10i, const double u11r, const double u11i) {

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int maxind = dim / 2;
	const unsigned int targetMaskm = targetMask - 1;
	while (i < maxind) {
		size_t t1 = (i&targetMaskm) ^ ((i&(~targetMaskm)) << 1);
		size_t t2 = t1^targetMask;
		double a1r = state[2 * t1];
		double a1i = state[2 * t1 + 1];
		double a2r = state[2 * t2];
		double a2i = state[2 * t2 + 1];
		state[2 * t1] = u00r * a1r - u00i * a1i + u01r * a2r - u01i * a2i;
		state[2 * t1 + 1] = u00i * a1r + u00r * a1i + u01i * a2r + u01r * a2i;
		state[2 * t2] = u10r * a1r - u10i * a1i + u11r * a2r - u11i * a2i;
		state[2 * t2 + 1] = u10i * a1r + u10r * a1i + u11i * a2r + u11r * a2i;
		i += blockDim.x * gridDim.x;
	}
}
void op_u(double* state, const size_t dim, const unsigned int target, const double u1, const double u2, const double u3) {
	const size_t targetMask = ((size_t)1) << target;
	double u00r, u01r, u10r, u11r, u00i, u01i, u10i, u11i;
	unsigned int blockCount, threadCount;

	u00r = cos((u2 + u3) / 2) * cos(u1 / 2);
	u00i = -sin((u2 + u3) / 2) * cos(u1 / 2);
	u01r = -cos((u2 - u3) / 2) * sin(u1 / 2);
	u01i = sin((u2 - u3) / 2) * sin(u1 / 2);
	u10r = cos((u2 - u3) / 2) * sin(u1 / 2);
	u10i = sin((u2 - u3) / 2) * sin(u1 / 2);
	u11r = cos((u2 + u3) / 2) * cos(u1 / 2);
	u11i = sin((u2 + u3) / 2) * cos(u1 / 2);

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_u << < blockCount, threadCount >> > (state, dim, targetMask, u00r, u00i, u01r, u01i, u10r, u10i, u11r, u11i);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_u failed : %s\n", hipGetErrorString(cudaStatus));
	}
}


/*
2qubit unitary operation
control not

"target" must be different from "control"
*/
__global__ void kernel_op_cx(double *state, const size_t dim, const size_t targetMask, const size_t controlMask) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int maxind = dim / 4;
	const unsigned int mask1 = (min(targetMask, controlMask)) - 1;
	const unsigned int mask2 = (max(targetMask, controlMask)) - 1;
	for (i = 0; i < maxind; i++) {
		size_t t1, t2;
		t1 = (i&mask1) ^ ((i&(~mask1)) << 1);
		t1 = (t1&mask2) ^ ((t1&(~mask2)) << 1) ^ controlMask;
		t2 = t1^targetMask;
		double a1r = state[2 * t1];
		double a1i = state[2 * t1 + 1];
		double a2r = state[2 * t2];
		double a2i = state[2 * t2 + 1];
		state[2 * t1] = a2r;
		state[2 * t1 + 1] = a2i;
		state[2 * t2] = a1r;
		state[2 * t2 + 1] = a1i;
		i += blockDim.x * gridDim.x;
	}
}
void op_cx(double* state, const size_t dim, const unsigned int target, const unsigned int control) {
	const size_t targetMask = ((size_t)1) << target;
	const size_t controlMask = ((size_t)1) << control;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_cx << < blockCount, threadCount >> > (state, dim, targetMask, controlMask);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_cx failed : %s\n", hipGetErrorString(cudaStatus));
	}
}


/*
1qubit non-unitary operation
post-select 0-outcome
*/
__global__ void kernel_op_post0(const double *state, double* nstate, const size_t dim, const size_t targetMask, const double norm) {
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		if ((i&targetMask) == 0) {
			nstate[2 * i] = state[2 * i] * norm;
			nstate[2 * i + 1] = state[2 * i + 1] * norm;
		}
		else {
			nstate[2 * i] = 0;
			nstate[2 * i + 1] = 0;
		}

		i += blockDim.x * gridDim.x;
	}
}

void op_post0(const double* state, double* nstate, const size_t dim, const unsigned int target, const double norm) {
	const size_t targetMask = ((size_t)1) << target;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_post0 << <blockCount, threadCount >> > (state, nstate, dim, targetMask, norm);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_post0 failed : %s\n", hipGetErrorString(cudaStatus));
	}
}

/*
1qubit non-unitary operation
post-select 1-outcome
*/
__global__ void kernel_op_post1(const double *state, double* nstate, const size_t dim, const size_t targetMask, const double norm) {

	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		if (i&targetMask) {
			nstate[2 * i] = state[2 * i] * norm;
			nstate[2 * i + 1] = state[2 * i + 1] * norm;
		}
		else {
			nstate[2 * i] = 0;
			nstate[2 * i + 1] = 0;
		}

		i += blockDim.x * gridDim.x;
	}
}
void op_post1(const double* state, double* nstate, const size_t dim, const unsigned int target, const double norm) {
	const size_t targetMask = ((size_t)1) << target;
	unsigned int blockCount, threadCount;

	threadCount = min((unsigned int)dim, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)dim / g_maxThreadsPerBlock, 1);

	kernel_op_post1 << <blockCount, threadCount >> > (state, nstate, dim, targetMask, norm);
	hipDeviceSynchronize();
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda op_post1 failed : %s\n", hipGetErrorString(cudaStatus));
	}
}

/*
1qubit non-unitary operation
measurement, and return outcome
*/
unsigned int op_meas(const double* state, double* nstate, const size_t dim, const unsigned int target) {
	double prob1;
	double randomValue;
	double norm;
	unsigned int outcome;

	prob1 = stat_prob1(state, nstate, dim, target);
	randomValue = rng();
	if (randomValue > prob1) {
		outcome = 0;
		norm = 1. / sqrt(1 - prob1);
		op_post0(state, nstate, dim, target, norm);
	}
	else {
		outcome = 1;
		norm = 1. / sqrt(prob1);
		op_post1(state, nstate, dim, target, norm);
	}
	return outcome;
}

/*
calculate probability with which we obtain outcome 1
*/
template <unsigned int blockSize>
__global__ void kernel_stat_prob1_optimized(const double *g_idata, double *g_odata, const size_t N, const size_t targetMask)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < N) {
		if((i/2)&targetMask)				
			sdata[tid] += g_idata[i] * g_idata[i];
		if(((i+blockSize)/2)&targetMask)	
			sdata[tid] += g_idata[i + blockSize] * g_idata[i + blockSize];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 1024){ if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
template <unsigned int blockSize>
__global__ void kernel_stat_sum_optimized(const double *g_idata, double *g_odata, const size_t N)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*(blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;
	while (i < N) {
		sdata[tid] += g_idata[i] + g_idata[i + blockSize];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void kernel_stat_prob1(const double *g_idata, double *g_odata, const size_t N, const size_t targetMask)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = ((i/2)&targetMask) ? g_idata[i] * g_idata[i] : 0.;
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
__global__ void kernel_stat_sum(const double *g_idata, double *g_odata, const size_t N)
{
	extern __shared__ double sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
	sdata[tid] = g_idata[i]; 
	__syncthreads();
	for (unsigned int s = blockDim.x / 2; s>0; s >>= 1) {
		if (tid < s) {
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads();
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

double stat_prob1(const double* state, double* workspace, const size_t dim, const unsigned int target) {
	const size_t targetMask = ((size_t)1) << target;
	double prob1 = 0.;
	unsigned int threadCount, blockCount, sharedMemSize, cursor, N;
	hipError_t cudaStatus;

	N = (unsigned int)dim * 2;
	threadCount = min(N, g_maxThreadsPerBlock);
	blockCount = max((unsigned int)N / (2*g_maxThreadsPerBlock), 1);
	sharedMemSize = threadCount * sizeof(double);

	// mapping squared values from state to workspace 
	if (threadCount == 1024 && blockCount>1)	kernel_stat_prob1_optimized<1024> << <blockCount, threadCount, sharedMemSize >> > (state, workspace, N, targetMask);
	else										kernel_stat_prob1 << <blockCount, threadCount, sharedMemSize >> > (state, workspace, N, targetMask);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "stat_prob1 fail : %s\n", hipGetErrorString(cudaStatus));
	}

	// accumulate workspace with length = blockCount
	cursor = 0;
	N = blockCount;
	while(N>1) {
		threadCount = min(N, g_maxThreadsPerBlock);
		blockCount = max((unsigned int)N / (2 * g_maxThreadsPerBlock), 1);
		sharedMemSize = threadCount * sizeof(double);

		double* fromPtr = workspace + cursor;
		double* toPtr = workspace + cursor + N;
		if (threadCount == 1024 && blockCount > 1)	kernel_stat_sum_optimized<1024> << <blockCount, threadCount, sharedMemSize >> > (fromPtr, toPtr , N);
		else										kernel_stat_sum << <blockCount, threadCount, sharedMemSize >> > (fromPtr, toPtr, N);
		hipDeviceSynchronize();
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "stat_prob1 loop fail : %s\n", hipGetErrorString(cudaStatus));
		}

		cursor += N;
		N = blockCount;
	}

	cudaStatus = hipMemcpy(&prob1, workspace + cursor, sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy fail : %s\n", hipGetErrorString(cudaStatus));
	}
	//printf("prob1 : %lf\n", prob1);
	return prob1;
}

void dump_vector(const double* state, const size_t dim, FILE* outStream) {
	size_t i; 
	double norm = 0.;
	double* local = (double*)malloc(2 * dim * sizeof(double));
	hipError_t cudaStatus;
	cudaStatus = hipMemcpy(local,state,2*dim*sizeof(double),hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed : %s\n", hipGetErrorString(cudaStatus));
	}
	else {
		for (i = 0; i < dim; i++) {
			fprintf(outStream, "%lld : %lf , %lf\n", i, local[2 * i], local[2 * i + 1]);
			norm += local[2 * i] * local[2 * i] + local[2 * i + 1] * local[2 * i + 1];
		}
		printf("norm :%lf\n", norm);
	}
	free(local);
}