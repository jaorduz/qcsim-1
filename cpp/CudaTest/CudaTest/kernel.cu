#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_complex.h"

#include <cstdio>
#include <cmath>
#include <cassert>
#include <random>
#include <algorithm>
#include <chrono>
#include <bitset>
#include <iostream>
#include <complex>
#include <conio.h>
#include <vector>
#include <fstream>
#include <string>

#define block 256
#define blockThread	1024

__global__ void IKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim,unsigned const int target)
{
	//int i = threadIdx.x;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	while (i < dim) {
		n[i] = o[i];
		i += blockDim.x*gridDim.x;
	}
}
__global__ void XKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int target)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << target;
	while (i < dim) {
		n[i] = o[i^shift];
		i += blockDim.x*gridDim.x;
	}
}
__global__ void YKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int target)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << target;
	while (i < dim) {
		int sign = 1 - ((i >> target) % 2) * 2;
		n[i] = hipCmul(
			make_hipDoubleComplex(0,sign),
			o[i^shift]
		);
		i += blockDim.x*gridDim.x;
	}
}
__global__ void ZKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int target)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << target;
	while (i < dim) {
		int sign = 1 - ((i >> target) % 2) * 2;
		n[i] = hipCmul(
			make_hipDoubleComplex(sign, 0),
			o[i]
		);
		i += blockDim.x*gridDim.x;
	}
}
__global__ void SKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int target)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << target;
	while (i < dim) {
		if ( (i>>target)%2) {
			n[i] = hipCmul(
				make_hipDoubleComplex(0, 1),
				o[i]
			);
		}
		else {
			n[i] = o[i];
		}
		i += blockDim.x*gridDim.x;
	}
}
__global__ void TKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int target)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << target;
	while (i < dim) {
		if ((i >> target) % 2) {
			n[i] = hipCmul(
				make_hipDoubleComplex(sqrt(0.5), sqrt(0.5)),
				o[i]
			);
		}
		else {
			n[i] = o[i];
		}
		i += blockDim.x*gridDim.x;
	}
}
__global__ void hadamardKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int k)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << k;
	while (i < dim) {
		int sign = 1 - ((i >> k) % 2) * 2;
		n[i] = hipCadd(
			hipCmul(make_hipDoubleComplex(sqrt(0.5), 0), o[i^shift]),
			hipCmul(make_hipDoubleComplex(sign*sqrt(0.5), 0), o[i])
		);
		i += blockDim.x*gridDim.x;
	}
}
__global__ void cnotKernel(hipDoubleComplex *n, const hipDoubleComplex *o, unsigned const int dim, unsigned const int k,unsigned const int l)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int shift = 1 << l;
	while (i < dim) {
		if ( (i>>k)%2 ) {
			n[i] = o[i^shift];
		}
		else {
			n[i] = o[i];
		}
		i += blockDim.x*gridDim.x;
	}
}

class MyCuda {
private:
	unsigned int _n;
	unsigned int _dim;
	hipDoubleComplex *stateLocal;
	hipDoubleComplex *stateOrg;
	hipDoubleComplex *stateNext;

public:
	MyCuda() :_n(0), _dim(0), stateOrg(0), stateNext(0),stateLocal(0)
	{
		hipError_t cudaStatus;
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			exit(0);
		}
	}
	virtual ~MyCuda() {
		hipError_t cudaStatus = hipDeviceReset();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceReset failed!");
		}
	}

	hipError_t init(int numQubit) {
		_n = numQubit;
		_dim = 1 << numQubit;

		hipError_t cudaStatus;
		stateLocal = new hipDoubleComplex[_dim];
		for (unsigned int i = 0; i < _dim; i++) stateLocal[i] = make_hipDoubleComplex(0,0);
		stateLocal[0] = make_hipDoubleComplex(1,0);

		cudaStatus = hipMalloc((void**)&stateOrg, _dim * sizeof(hipDoubleComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&stateNext, _dim * sizeof(hipDoubleComplex));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}

		cudaStatus = hipMemcpy(stateOrg, stateLocal, _dim * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		return cudaStatus;
	Error:
		hipFree(stateOrg);
		hipFree(stateNext);
		delete[] stateLocal;
		return cudaStatus;
	}

	void apply1QG(void(*kernel)(hipDoubleComplex*, const hipDoubleComplex*,const unsigned int, const unsigned int), int target) {
		assert(0 <= target && target < _n);
		// Launch a kernel on the GPU with one thread for each element.
		// <<block, thread per block>>
		// max thread per block = 2**10
		// max block = 
		if (_dim <= (1 << 10)) {
			kernel << <1, _dim >> >(stateNext, stateOrg, _dim, target);
		}
		else {

			//kernel << <min(256,_dim/1024), 1024 >> >(stateNext, stateOrg, _dim, target);
			kernel << <min(block,_dim/blockThread), blockThread >> >(stateNext, stateOrg, _dim, target);
		}

		hipDoubleComplex* stateTemp = stateOrg;
		stateOrg = stateNext;
		stateNext = stateOrg;

		/*
		// Check for any errors launching the kernel
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "apply kernel of single qubit operation launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		return cudaStatus;
		*/
	}
	void apply2QG(void(*kernel)(hipDoubleComplex*, const hipDoubleComplex*, const unsigned int, const unsigned int, const unsigned int), int control, int target) {
		assert(0 <= target && target < _n);
		// Launch a kernel on the GPU with one thread for each element.
		// <<block, thread per block>>
		// max thread per block = 2**10
		// max block = 
		if (_dim <= (1 << 10)) {
			kernel << <1, _dim >> >(stateNext, stateOrg, _dim, control, target);
		}
		else {

			//kernel << <min(256,_dim/1024), 1024 >> >(stateNext, stateOrg, _dim, target);
			kernel << <block, blockThread >> >(stateNext, stateOrg, _dim, control, target);
		}

		hipDoubleComplex* stateTemp = stateOrg;
		stateOrg = stateNext;
		stateNext = stateOrg;

		/*
		// Check for any errors launching the kernel
		hipError_t cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "apply kernel of two qubit operation launch failed: %s\n", hipGetErrorString(cudaStatus));
		}
		return cudaStatus;
		*/
	}

	hipError_t sync() {
		hipError_t cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			return cudaStatus;
		}
	}

	hipError_t getStatus() {
		// Copy output vector from GPU buffer to host memory.
		hipError_t cudaStatus = hipMemcpy(stateLocal, stateOrg, _dim * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		return cudaStatus;
	}

	void dumpState() {
		getStatus();
		std::cout << "dump" << std::endl;
		double norm = 0;
		for (unsigned int i = 0; i < _dim; i++) { 
			std::complex<double> val = std::complex<double>(stateLocal[i].x, stateLocal[i].y);
			norm += std::pow(std::abs(val),2);
			if (std::abs(val) != 0) {
				std::cout << val << "|";
				for (unsigned int j = 0; j < _n; j++) {
					std::cout << ((i >> j) % 2);
				}
				std::cout << "> ";
			}
		}
		std::cout << std::endl;
		std::cout << norm << std::endl;
		if (fabs(norm-1.0)>1e-8) {
			_getch();
		}
	}

	void close() {
		sync();
		hipFree(stateOrg);
		hipFree(stateNext);
		delete[] stateLocal;
	}
};

int test() {
	unsigned int n = 2;
	MyCuda* mc = new MyCuda();
	mc->init(n);
	mc->dumpState();
	mc->apply1QG(YKernel, 0);
	mc->dumpState();
	mc->apply1QG(XKernel, 0);
	mc->dumpState();
	mc->apply1QG(YKernel, 0);
	mc->dumpState();
	mc->apply1QG(hadamardKernel, 0);
	mc->dumpState();
	mc->apply2QG(cnotKernel, 0,1);
	mc->dumpState();
	mc->apply1QG(TKernel, 0);
	mc->dumpState();
	mc->close();
	delete mc;
	return 0;
}

std::vector<__int64> randomCircuitOneshot(unsigned int n, unsigned int depth) {
	std::mt19937 mt(0);
	std::vector<__int64> dur;

	MyCuda* mc = new MyCuda();
	mc->init(n);
	std::fstream ofs("gputime.txt", std::ios::app);
	ofs << n << " ";
	ofs.close();
	for (int d = 0; d < depth; d++) {
		auto start = std::chrono::system_clock::now();
		for (int i = 0; i < n; i++) {
			int r = mt() % 5;
			if (d == 0 && i == 0) r = 4;
			if (r == 0)			mc->apply1QG(XKernel, i);
			else if (r == 1)	mc->apply1QG(TKernel, i);
			else if (r==2)		mc->apply1QG(YKernel, i);
			else if(r==3)		mc->apply1QG(hadamardKernel, i);
			else if (r == 4) {
				if (i + 1 < n) {
					mc->apply2QG(cnotKernel, i, i+1);
					i++;
				}
				else {
					mc->apply1QG(SKernel,i);
				}
			}
		}
		mc->sync();
		__int64 time = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::system_clock::now() - start).count();
		dur.push_back(time);
		std::cout << d << " " << time << std::endl;

		std::fstream ofsa("gputime.txt", std::ios::app);
		ofsa << time << " ";
		ofsa.close();
	}
	mc->close();
	std::fstream ofse("gputime.txt", std::ios::app);
	ofse << std::endl;
	ofse.close();
	return dur;
}


int main(int argc, char** argv) {
	int n = 27;
	int depth = 100;
	if (argc > 1) {
		n = atoi(argv[1]);
		depth = atoi(argv[2]);
	}
	auto time = randomCircuitOneshot(n,depth);
	__int64 sum = 0;
	for (int i = 0; i < time.size(); i++) {
		sum += time[i];
	}
	std::cout << sum / time.size() << std::endl;

	return 0;
}